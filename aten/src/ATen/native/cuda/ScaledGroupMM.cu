#define TORCH_ASSERT_ONLY_METHOD_OPERATORS
#include <ATen/Dispatch.h>
#include <ATen/core/Tensor.h>
#include <ATen/cuda/HIPContext.h>
#include <ATen/cuda/nvrtc_stub/ATenNVRTC.h>
#include <c10/macros/Macros.h>

// Two warninngs in Cutlass included header files
C10_DIAGNOSTIC_PUSH_AND_IGNORED_IF_DEFINED("-Wset-but-not-used")
C10_DIAGNOSTIC_PUSH_AND_IGNORED_IF_DEFINED("-Wunused-but-set-parameter")

// Determine if the architecture supports rowwise scaled mm
// Currently failing on windows with:
// https://github.com/NVIDIA/cutlass/issues/1571
#if !defined(USE_ROCM) && !defined(_WIN32) && defined(CUDA_VERSION) && CUDA_VERSION >= 12000

#define BUILD_ROWWISE_FP8_KERNEL
#endif

#if defined(BUILD_ROWWISE_FP8_KERNEL)

#include <cute/tensor.hpp>
#include <cutlass/core_io.h>
#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/half.h>
#include <cutlass/numeric_types.h>
#include <cutlass/trace.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/version.h>

#include <cutlass/gemm/collective/collective_builder.hpp>
#include <cutlass/gemm/device/gemm_universal.h>
#include <cutlass/gemm/device/gemm_universal_adapter.h>
#include <cutlass/gemm/kernel/default_gemm_universal_with_visitor.h>
#include <cutlass/epilogue/collective/collective_builder.hpp>
#include <cutlass/epilogue/threadblock/fusion/visitors.hpp>

#include <cute/atom/mma_atom.hpp>
#include <cutlass/gemm/dispatch_policy.hpp>
#include <cutlass/gemm/kernel/gemm_universal.hpp>
#include <cutlass/util/packed_stride.hpp>

C10_DIAGNOSTIC_POP()
C10_DIAGNOSTIC_POP()

namespace {

constexpr int kNumSMsForH100 = 132;
}

#endif

namespace at::cuda::detail {

void f8f8bf16_grouped_mm(
    at::Tensor mat_a, // FP8
    at::Tensor mat_b, // FP8
    at::Tensor scale_a, // FP32
    at::Tensor scale_b, // FP32
    std::optional<at::Tensor> offs_a, 
    std::optional<at::Tensor> offs_b, 
    std::optional<at::Tensor> bias, // BF16
    bool use_fast_accum,
    at::Tensor& out) {
#if defined(BUILD_ROWWISE_FP8_KERNEL)
#else
#endif     
    }

} // namespace at::cuda::detail